#define GRB_USE_CUDA
#define private public

#include <iostream>
#include <algorithm>
#include <string>

#include <cstdio>
#include <cstdlib>

// #include <hip/hip_runtime_api.h>
#include <boost/program_options.hpp>

#include "graphblas/graphblas.hpp"
#include "graphblas/algorithm/sssp.hpp"
#include "graphblas/algorithm/common.hpp"
#include "test/test.hpp"

bool debug_;
bool memory_;

int main(int argc, char** argv) {
  std::vector<graphblas::Index> row_indices;
  std::vector<graphblas::Index> col_indices;
  std::vector<float> values;
  graphblas::Index nrows, ncols, nvals;

  // Parse arguments
  bool debug;
  bool transpose;
  bool mtxinfo;
  int  directed;
  int  niter;
  int  source;
  int  seed;
  char* dat_name;
  po::variables_map vm;

  // Read in sparse matrix
  if (argc < 2) {
    fprintf(stderr, "Usage: %s [matrix-market-filename]\n", argv[0]);
    exit(1);
  } else {
    parseArgs(argc, argv, &vm);
    debug     = vm["debug"    ].as<bool>();
    transpose = vm["transpose"].as<bool>();
    mtxinfo   = vm["mtxinfo"  ].as<bool>();
    directed  = vm["directed" ].as<int>();
    niter     = vm["niter"    ].as<int>();
    source    = vm["source"   ].as<int>();
    seed      = vm["seed"     ].as<int>();

    /*!
     * This is an imperfect solution, because this should happen in
     * desc.loadArgs(vm) instead of application code!
     * TODO(@ctcyang): fix this
     */
    readMtx(argv[argc-1], &row_indices, &col_indices, &values, &nrows, &ncols,
        &nvals, directed, mtxinfo, &dat_name);
  }

  // Descriptor desc
  graphblas::Descriptor desc;
  CHECK(desc.loadArgs(vm));
  if (transpose)
    CHECK(desc.toggle(graphblas::GrB_INP1));

  // Matrix A
  graphblas::Matrix<float> a(nrows, ncols);
  CHECK(a.build(&row_indices, &col_indices, &values, nvals, GrB_NULL,
      dat_name));
  CHECK(a.nrows(&nrows));
  CHECK(a.ncols(&ncols));
  CHECK(a.nvals(&nvals));
  if (debug) CHECK(a.print());

  // For SSSP, do uniform random distance between 1 and 64
  // Set seed
  setEnv("GRB_SEED", seed);
  setEnv("GRB_UNIFORM_START", 1);
  setEnv("GRB_UNIFORM_END", 64);
  desc.set(GrB_BACKEND, GrB_SEQUENTIAL);
  graphblas::apply<float, float, float>(&a, GrB_NULL, GrB_NULL, set_uniform_random<float>(), &a, &desc);
  desc.set(GrB_BACKEND, GrB_CUDA);
  if (debug) CHECK(a.print());

  // Vector v
  graphblas::Vector<float> v(nrows);

  // Cpu BFS
  CpuTimer sssp_cpu;
  float* h_sssp_cpu = reinterpret_cast<float*>(malloc(nrows*sizeof(float)));
  int depth = 10000;
  sssp_cpu.Start();
  int d = graphblas::algorithm::ssspCpu(source, &a, h_sssp_cpu, depth,
      transpose);
  sssp_cpu.Stop();

  // Warmup
  CpuTimer warmup;
  warmup.Start();
  graphblas::algorithm::sssp(&v, &a, source, &desc);
  warmup.Stop();

  std::vector<float> h_sssp_gpu;
  CHECK(v.extractTuples(&h_sssp_gpu, &nrows));
  VERIFY_LIST_FLOAT(h_sssp_cpu, h_sssp_gpu, nrows);

  // Benchmark
  graphblas::Vector<float> y(nrows);
  CpuTimer vxm_gpu;
  // hipProfilerStart();
  vxm_gpu.Start();
  float tight = 0.f;
  float val;
  for (int i = 0; i < niter; i++) {
    val = graphblas::algorithm::sssp(&y, &a, source, &desc);
    tight += val;
  }
  // hipProfilerStop();
  vxm_gpu.Stop();

  float flop = 0;
  std::cout << "cpu, " << sssp_cpu.ElapsedMillis() << ", \n";
  std::cout << "warmup, " << warmup.ElapsedMillis() << ", " <<
    flop/warmup.ElapsedMillis()/1000000.0 << "\n";
  float elapsed_vxm = vxm_gpu.ElapsedMillis();
  std::cout << "tight, " << tight/niter << "\n";
  std::cout << "vxm, " << elapsed_vxm/niter << "\n";

  if (niter) {
    std::vector<float> h_sssp_gpu2;
    CHECK(y.extractTuples(&h_sssp_gpu2, &nrows));
    VERIFY_LIST_FLOAT(h_sssp_cpu, h_sssp_gpu2, nrows);
  }

  return 0;
}
