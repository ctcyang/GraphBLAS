#define GRB_USE_CUDA
#define private public

#include <iostream>
#include <algorithm>
#include <string>

#include <cstdio>
#include <cstdlib>

// #include <hip/hip_runtime_api.h>

#include <boost/program_options.hpp>

#include "graphblas/graphblas.hpp"
#include "graphblas/algorithm/bfs.hpp"
#include "test/test.hpp"

bool debug_;
bool memory_;

int main(int argc, char** argv) {
  std::vector<graphblas::Index> row_indices;
  std::vector<graphblas::Index> col_indices;
  std::vector<float> values;
  graphblas::Index nrows, ncols, nvals;

  // Parse arguments
  bool debug;
  bool transpose;
  bool mtxinfo;
  int  directed;
  int  niter;
  int  source;
  char* dat_name;
  po::variables_map vm;

  // Read in sparse matrix
  if (argc < 2) {
    fprintf(stderr, "Usage: %s [matrix-market-filename]\n", argv[0]);
    exit(1);
  } else {
    parseArgs(argc, argv, &vm);
    debug     = vm["debug"    ].as<bool>();
    transpose = vm["transpose"].as<bool>();
    mtxinfo   = vm["mtxinfo"  ].as<bool>();
    directed  = vm["directed" ].as<int>();
    niter     = vm["niter"    ].as<int>();
    source    = vm["source"   ].as<int>();

    // This is an imperfect solution, because this should happen in
    // desc.loadArgs(vm) instead of application code!
    // TODO(@ctcyang): fix this
    readMtx(argv[argc-1], &row_indices, &col_indices, &values, &nrows, &ncols,
        &nvals, directed, mtxinfo, &dat_name);
  }

  // Descriptor desc
  graphblas::Descriptor desc;
  CHECK(desc.loadArgs(vm));
  if (transpose)
    CHECK(desc.toggle(graphblas::GrB_INP1));

  // Matrix A
  graphblas::Matrix<float> a(nrows, ncols);
  CHECK(a.build(&row_indices, &col_indices, &values, nvals, GrB_NULL,
      dat_name));
  CHECK(a.nrows(&nrows));
  CHECK(a.ncols(&ncols));
  CHECK(a.nvals(&nvals));
  if (debug) CHECK(a.print());

  // Vector v
  graphblas::Vector<float> v(nrows);

  // Cpu BFS
  CpuTimer bfs_cpu;
  graphblas::Index* h_bfs_cpu = reinterpret_cast<graphblas::Index*>(
      malloc(nrows*sizeof(graphblas::Index)));
  int depth = 10000;
  bfs_cpu.Start();
  int d = graphblas::algorithm::bfsCpu(source, &a, h_bfs_cpu, depth, transpose);
  bfs_cpu.Stop();

  // Warmup
  CpuTimer warmup;
  warmup.Start();
  graphblas::algorithm::bfs(&v, &a, source, &desc);
  warmup.Stop();

  std::vector<float> h_bfs_gpu;
  CHECK(v.extractTuples(&h_bfs_gpu, &nrows));
  VERIFY_LIST(h_bfs_cpu, h_bfs_gpu, nrows);

  // Benchmark
  graphblas::Vector<float> y(nrows);
  CpuTimer vxm_gpu;
  // hipProfilerStart();
  vxm_gpu.Start();
  float tight = 0.f;
  float val;
  for (int i = 0; i < niter; i++) {
    val = graphblas::algorithm::bfs(&y, &a, source, &desc);
    tight += val;
  }
  // hipProfilerStop();
  vxm_gpu.Stop();

  float flop = 0;
  std::cout << "cpu, " << bfs_cpu.ElapsedMillis() << ", \n";
  std::cout << "warmup, " << warmup.ElapsedMillis() << ", " <<
    flop/warmup.ElapsedMillis()/1000000.0 << "\n";
  float elapsed_vxm = vxm_gpu.ElapsedMillis();
  std::cout << "tight, " << tight/niter << "\n";
  std::cout << "vxm, " << elapsed_vxm/niter << "\n";

  if (niter) {
    std::vector<float> h_bfs_gpu2;
    CHECK(y.extractTuples(&h_bfs_gpu2, &nrows));
    VERIFY_LIST(h_bfs_cpu, h_bfs_gpu2, nrows);
  }

  return 0;
}
